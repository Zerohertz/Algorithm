#include <hip/hip_runtime.h>

__global__ void vector_add(const float *A, const float *B, float *C, int N)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float *A, const float *B, float *C, int N)
{
    int threadsPerBlock = 256;
    int blocksPerGrid   = (N + threadsPerBlock - 1) / threadsPerBlock;

    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}
